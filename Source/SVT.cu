// nvcc SVT.cu -o svt -lglut -lm -lGLU -lGL

/*
 This file contains all the the main controller functions that setup the simulation, then run and manage the simulation.
 The functions are listed below in the order they appear.
 
 void nBody(float);
 void setupCudaInvironment();
 void readSimulationParameters();
 void setup();
 int main(int, char**);
*/

// Local include files
#include "./header.h"
#include "./setNodesAndMuscles.h"
#include "./callBackFunctions.h"
#include "./viewDrawAndTerminalFunctions.h"
#include "./cudaFunctions.h"

/*
 This function is called by the openGL idle function. Hense this function is call every time openGL is not doing anything else,
 which is most of the time.
 This function orchstracts the simulation by;
 1: Calling the getForces function which gets all the forces except the drag force on all nodes.
 2: Calling the upDateNodes function which moves the nodes based off of the forces from the getForces function.
    It uses the leap-frog formulas to integrate the nodes forward in time. It also sees if a node is a beat node  
    and if it needs to send out a segnal.
 3: Calling the updateMuscles function to adjust where they are in their cycle and react acordingly.
 4: Sees if it is time to recenter the simulation.
 5: Sees if simulation needs to be redrawn to the screen.
 6: Sees if the terminal screen needs to be updated.
 
 Note: If Pause is on it skips all this and if Contraction is not on it skips all of its moving calculations
 and only performs calculations that deal with electrical conduction and muscle timing. 
*/
void nBody(float dt)
{	
	if(PauseIs == false)
	{	
		if(ContractionIsOn == true)
		{
			getForces<<<GridNodes, BlockNodes>>>(MuscleGPU, NodeGPU, dt, NumberOfNodes, CenterOfSimulation, MuscleCompresionStopFraction, RadiusOfLeftAtrium, DiastolicPressureLA, SystolicPressureLA);
			cudaErrorCheck(__FILE__, __LINE__);
			hipDeviceSynchronize();
		}
		updateNodes<<<GridNodes, BlockNodes>>>(NodeGPU, NumberOfNodes, MUSCLES_PER_NODE, MuscleGPU, Drag, dt, RunTime, ContractionIsOn);
		cudaErrorCheck(__FILE__, __LINE__);
		hipDeviceSynchronize();
		updateMuscles<<<GridMuscles, BlockMuscles>>>(MuscleGPU, NodeGPU, NumberOfMuscles, NumberOfNodes, dt, ReadyColor, ContractingColor, RestingColor, RelativeColor);
		cudaErrorCheck(__FILE__, __LINE__);
		hipDeviceSynchronize();
		
		if(ContractionIsOn == true)
		{
			RecenterCount++;
			if(RecenterCount == RecenterRate) 
			{
				recenter<<<1, BLOCKCENTEROFMASS>>>(NodeGPU, NumberOfNodes, MassOfLeftAtrium, CenterOfSimulation);
				cudaErrorCheck(__FILE__, __LINE__);
				RecenterCount = 0;
			}
		}
		
		DrawTimer++;
		if(DrawTimer == DrawRate) 
		{
			copyNodesMusclesFromGPU();
			drawPicture();
			DrawTimer = 0;
		}
		
		PrintTimer += dt;
		if(PrintRate <= PrintTimer) 
		{
			terminalPrint();
			PrintTimer = 0.0;
		}
		
		RunTime += dt; 
	}
	else
	{
		drawPicture();
	}
}

/*
 Setting up the CUDA invironment. We have three:
 1: Node based
 2: Muscle based
 3: Just one block used for recentering the simulation.
*/
void setupCudaInvironment()
{
	// 1:
	BlockNodes.x = BLOCKNODES;
	BlockNodes.y = 1;
	BlockNodes.z = 1;
	
	GridNodes.x = (NumberOfNodes - 1)/BlockNodes.x + 1;
	GridNodes.y = 1;
	GridNodes.z = 1;
	
	// 2:
	BlockMuscles.x = BLOCKMUSCLES;
	BlockMuscles.y = 1;
	BlockMuscles.z = 1;
	
	GridMuscles.x = (NumberOfMuscles - 1)/BlockMuscles.x + 1;
	GridMuscles.y = 1;
	GridMuscles.z = 1;
	
	// 3:
	if((BLOCKCENTEROFMASS > 0) && (BLOCKCENTEROFMASS & (BLOCKCENTEROFMASS - 1)) != 0) 
	{
        	printf("\nBLOCKCENTEROFMASS = %d. This is not a power of 2.", BLOCKCENTEROFMASS);
        	printf("\nBLOCKCENTEROFMASS must be a power of 2 for the center of mass reduction to work.");
        	printf("\nFix this number in the header.h file and try again.");
        	printf("\nGood Bye.\n");
        	exit(0);
        }
}

/*
 This function reads in all the user defined parameters in the simulationSetup file.
*/
void readSimulationParameters()
{
	ifstream data;
	string name;
	
	data.open("./simulationSetup");
	
	if(data.is_open() == 1)
	{
		getline(data,name,'=');
		data >> NodesMusclesFileOrPreviousRunsFile;
		
		getline(data,name,'=');
		data >> NodesMusclesFileName;
		
		getline(data,name,'=');
		data >> PreviousRunFileName;
		
		getline(data,name,'=');
		data >> LineWidth;
		
		getline(data,name,'=');
		data >> NodeRadiusAdjustment;
		
		getline(data,name,'=');
		data >> MyocyteForcePerMass;
		
		getline(data,name,'=');
		data >> MyocyteForcePerMassMultiplier;
		
		getline(data,name,'=');
		data >> MyocyteForcePerMassSTD;
		
		getline(data,name,'=');
		data >> DiastolicPressureLA;
		
		getline(data,name,'=');
		data >> SystolicPressureLA;
		
		getline(data,name,'=');
		data >> PressureMultiplier;
		
		getline(data,name,'=');
		data >> MassOfLeftAtrium;
		
		getline(data,name,'=');
		data >> RadiusOfLeftAtrium;
		
		getline(data,name,'=');
		data >> Drag;
		
		getline(data,name,'=');
		data >> ContractionIsOn;
		
		getline(data,name,'=');
		data >> MuscleRelaxedStrengthFraction;
		
		getline(data,name,'=');
		data >> MuscleCompresionStopFraction;
		
		getline(data,name,'=');
		data >> MuscleCompresionStopFractionSTD;
		
		getline(data,name,'=');
		data >> BaseMuscleRefractoryPeriod;
		
		getline(data,name,'=');
		data >> MuscleRefractoryPeriodSTD;
		        
		getline(data,name,'=');
		data >> BaseAbsoluteRefractoryPeriodFraction;
		
		getline(data,name,'=');
		data >> AbsoluteRefractoryPeriodFractionSTD;
		
		getline(data,name,'=');
		data >> BaseMuscleConductionVelocity;
		
		getline(data,name,'=');
		data >> MuscleConductionVelocitySTD;
		
		getline(data,name,'=');
		data >> BeatPeriod;
		
		getline(data,name,'=');
		data >> PrintRate;
		
		getline(data,name,'=');
		data >> DrawRate;
		
		getline(data,name,'=');
		data >> Dt;
		
		getline(data,name,'=');
		data >> ReadyColor.x;
		
		getline(data,name,'=');
		data >> ReadyColor.y;
		
		getline(data,name,'=');
		data >> ReadyColor.z;
		
		getline(data,name,'=');
		data >> ContractingColor.x;
		
		getline(data,name,'=');
		data >> ContractingColor.y;
		
		getline(data,name,'=');
		data >> ContractingColor.z;
		
		getline(data,name,'=');
		data >> RestingColor.x;
		
		getline(data,name,'=');
		data >> RestingColor.y;
		
		getline(data,name,'=');
		data >> RestingColor.z;
		
		getline(data,name,'=');
		data >> RelativeColor.x;
		
		getline(data,name,'=');
		data >> RelativeColor.y;
		
		getline(data,name,'=');
		data >> RelativeColor.z;
		
		getline(data,name,'=');
		data >> DeadColor.x;
		
		getline(data,name,'=');
		data >> DeadColor.y;
		
		getline(data,name,'=');
		data >> DeadColor.z;
		
		getline(data,name,'=');
		data >> BackGround.x;
		
		getline(data,name,'=');
		data >> BackGround.y;
		
		getline(data,name,'=');
		data >> BackGround.z;
	}
	else
	{
		printf("\nTSU Error could not open simulationSetup file\n");
		exit(0);
	}
	
	data.close();
	printf("\n Simulation Parameters have been read in.");
}

/*
 This function calls all the functions that are used to setup the nodes muscles and initial prameters 
 of the simulation.
*/
void setup()
{	
	// Seading the random number generater.
	time_t t;
	srand((unsigned) time(&t));
	
	// Getting user inputs.
	readSimulationParameters();
	
	// Getting nodes and muscle from blender gererated files or a previous run file.
	if(NodesMusclesFileOrPreviousRunsFile == 0)
	{
		setNodesFromBlenderFile();
		checkNodes();
		setMusclesFromBlenderFile();
		linkNodesToMuscles();
		setRemainingNodeAndMuscleAttributes();
		hardCodedAblations();
		hardCodedPeriodicEctopicEvents();
		hardCodedIndividualMuscleAttributes();
		for(int i = 0; i < NumberOfMuscles; i++)
		{	
			checkMuscle(i);
		}
	}
	else if(NodesMusclesFileOrPreviousRunsFile == 1)
	{
		getNodesandMusclesFromPreviuosRun();
	}
	else
	{
		printf("\n Bad NodesMusclesFileOrPreviousRunsFile type.");
		printf("\n Good Bye.");
		exit(0);
	}
	
	// Setting parameters that are not initially read from the node and muscle or previous run file.
	setRemainingParameters();
	
	// Setting up the CUDA parallel structure to be used.
	setupCudaInvironment();
	
	// Sending all the info that we have just created to the GPU so it can start crunching numbers.
	copyNodesMusclesToGPU();
        
	printf("\n");
	char temp;
	printf("\033[0;31m");
	printf("\n\n The simulation has not been started.");
	printf("\n Hit any key and return to begin.\n\n");
	printf("\033[0m");
	scanf("%s", &temp); 
	
	terminalPrint();
}

/*
 In main we mostly just setup the openGL invironment and kickoff the glutMainLoop function.
*/
int main(int argc, char** argv)
{
	setup();
	
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	Far = 80.0*RadiusOfLeftAtrium;

	//Direction here your eye is located location
	EyeX = 0.0*RadiusOfLeftAtrium;
	EyeY = 0.0*RadiusOfLeftAtrium;
	EyeZ = 2.0*RadiusOfLeftAtrium;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;

	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("SVT");
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	glClearColor(BackGround.x, BackGround.y, BackGround.z, 0.0);
	
	//GLfloat light_position[] = {EyeX, EyeY, EyeZ, 0.0};
	GLfloat light_position[] = {1.0, 1.0, 1.0, 1.0}; //where the light is: {x,y,z,w}, w=0.0 is infinite light aiming at x,y,z, w=1.0 is a point light radiating from x,y,z
	GLfloat light_ambient[]  = {1.0, 1.0, 1.0, 1.0}; //what color is the ambient light, {r,g,b,a}, a= opacity 1.0 is fully visible, 0.0 is invisible
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0}; //does light reflect off of the object, {r,g,b,a}, a has no effect
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0}; //does light highlight shiny surfaces, {r,g,b,a}. i.e what light reflects to viewer
	GLfloat lmodel_ambient[] = {1.0, 1.0, 1.0, 1.0}; //global ambient light, {r,g,b,a}, applies uniformly to all objects in the scene
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0}; //reflective properties of an object, {r,g,b,a}, highlights are currently white
	GLfloat mat_shininess[]  = {128.0}; //how shiny is the surface of an object, 0.0 is dull, 128.0 is very shiny
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);

	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	//glutMouseFunc(mouseWheelCallback);
	//glutMouseWheelFunc(mouseWheelCallback);
	//glutMotionFunc(mouseMotionCallback);
    	glutPassiveMotionFunc(mousePassiveMotionCallback);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMouseFunc(mymouse);
	glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	glutSetCursor(GLUT_CURSOR_DESTROY);
	glEnable(GL_DEPTH_TEST);
	
	glutMainLoop();
	return 0;
}
